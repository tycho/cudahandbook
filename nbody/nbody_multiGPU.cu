#include "hip/hip_runtime.h"
/*
 *
 * nbody_multiGPU.cu
 *
 * Single-threaded multi-GPU implementation of the O(N^2) N-body calculation.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in
 *    the documentation and/or other materials provided with the
 *    distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include <chError.h>
#include <chTimer.h>
#include <chThread.h>

#include "nbody.h"
#include "nbody_multiGPU_shared.cuh"

#include "bodybodyInteraction.cuh"

using namespace cudahandbook::threading;

__global__ void
ComputeNBodyGravitation_multiGPU(
    float *force,
    float const * const posMass,
    float softeningSquared,
    size_t base,
    size_t n,
    size_t N )
{
    ComputeNBodyGravitation_Shared_multiGPU(
        force,
        posMass,
        softeningSquared,
        base,
        n,
        N );
}

float
ComputeGravitation_multiGPU(
    float *force,
    float const * const posMass,
    float softeningSquared,
    size_t N
)
{
    hipError_t status;

    float ret = 0.0f;

    float *dptrPosMass[g_maxGPUs];
    float *dptrForce[g_maxGPUs];
    int oldDevice;

    chTimerTimestamp start, end;
    chTimerGetTime( &start );

    memset( dptrPosMass, 0, sizeof(dptrPosMass) );
    memset( dptrForce, 0, sizeof(dptrForce) );
    size_t bodiesPerGPU = N / g_numGPUs;
    if ( (0 != N % g_numGPUs) || (g_numGPUs > g_maxGPUs) ) {
        return 0.0f;
    }
    CUDART_CHECK( hipGetDevice( &oldDevice ) );

    // kick off the asynchronous memcpy's - overlap GPUs pulling
    // host memory with the CPU time needed to do the memory
    // allocations.
    for ( size_t i = 0; i < g_numGPUs; i++ ) {
        CUDART_CHECK( hipSetDevice( i ) );
        CUDART_CHECK( hipMalloc( &dptrPosMass[i], 4*N*sizeof(float) ) );
        CUDART_CHECK( hipMalloc( &dptrForce[i], 3*bodiesPerGPU*sizeof(float) ) );
        CUDART_CHECK( hipMemcpyAsync(
            dptrPosMass[i],
            g_hostAOS_PosMass,
            4*N*sizeof(float),
            hipMemcpyHostToDevice ) );
    }
    for ( size_t i = 0; i < g_numGPUs; i++ ) {
        CUDART_CHECK( hipSetDevice( i ) );
        ComputeNBodyGravitation_multiGPU<<<300,256,256*sizeof(float4)>>>(
            dptrForce[i],
            dptrPosMass[i],
            softeningSquared,
            i*bodiesPerGPU,
            bodiesPerGPU,
            N );
        CUDART_CHECK( hipMemcpyAsync(
            g_hostAOS_Force+3*bodiesPerGPU*i,
            dptrForce[i],
            3*bodiesPerGPU*sizeof(float),
            hipMemcpyDeviceToHost ) );
    }
    // Synchronize with each GPU in turn.
    for ( size_t i = 0; i < g_numGPUs; i++ ) {
        CUDART_CHECK( hipSetDevice( i ) );
        CUDART_CHECK( hipDeviceSynchronize() );
    }
    chTimerGetTime( &end );
    ret = chTimerElapsedTime( &start, &end ) * 1000.0f;
Error:
    for ( size_t i = 0; i < g_numGPUs; i++ ) {
        hipFree( dptrPosMass[i] );
        hipFree( dptrForce[i] );
    }
    hipSetDevice( oldDevice );
    return ret;
}

/* vim: set ts=4 sts=4 sw=4 et: */
