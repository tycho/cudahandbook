#include "hip/hip_runtime.h"
/*
 *
 * nbody.cu
 *
 * N-body example that illustrates gravitational simulation.
 * This is the type of computation that GPUs excel at:
 * parallelizable, with lots of FLOPS per unit of external
 * memory bandwidth required.
 *
 * Requires: No minimum SM requirement.  If SM 3.x is not available,
 * this application quietly replaces the shuffle and fast-atomic
 * implementations with the shared memory implementation.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in
 *    the documentation and/or other materials provided with the
 *    distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>

// for kbhit()
#include <ch_conio.h>

#include <math.h>

#include <chError.h>
#include <chThread.h>
#include <chTimer.h>

#include "nbody.h"

#include "bodybodyInteraction.cuh"

using namespace cudahandbook::threading;

enum nbodyAlgorithm_enum {
    CPU_AOS = 0,    /* This is the golden implementation */
    CPU_AOS_tiled,
    CPU_SOA,
    CPU_SOA_tiled,
#ifdef HAVE_SIMD
    CPU_SIMD,
#endif
    GPU_AOS,
    GPU_Shared,
    GPU_Const,
    multiGPU,
// SM 3.0 only
    GPU_Shuffle,
    GPU_AOS_tiled,
    GPU_AOS_tiled_const,
//    GPU_Atomic
};

static const char *rgszAlgorithmNames[] = {
    "CPU_AOS",
    "CPU_AOS_tiled",
    "CPU_SOA",
    "CPU_SOA_tiled",
#ifdef HAVE_SIMD
    "CPU_SIMD",
#endif
    "GPU_AOS",
    "GPU_Shared",
    "GPU_Const",
    "multiGPU",
// SM 3.0 only
    "GPU_Shuffle",
    "GPU_AOS_tiled",
    "GPU_AOS_tiled_const",
//    "GPU_Atomic"
};

static inline void
randomVector( float v[3] )
{
    float lenSqr;
    do {
        v[0] = rand() / (float) RAND_MAX * 2 - 1;
        v[1] = rand() / (float) RAND_MAX * 2 - 1;
        v[2] = rand() / (float) RAND_MAX * 2 - 1;
        lenSqr = v[0]*v[0]+v[1]*v[1]+v[2]*v[2];
    } while ( lenSqr > 1.0f );
}

static void
randomUnitBodies( float *pos, float *vel, size_t N )
{
    for ( size_t i = 0; i < N; i++ ) {
        randomVector( &pos[4*i] );
        randomVector( &vel[4*i] );
        pos[4*i+3] = 1.0f;  // unit mass
        vel[4*i+3] = 1.0f;
    }
}

template<typename T>
static float
relError( float a, float b )
{
    if ( a == b ) return 0.0f;
    return fabsf(a-b)/b;
}

static bool g_bCUDAPresent;
static bool g_bSM30Present;

float *g_hostAOS_PosMass;
float *g_hostAOS_VelInvMass;
float *g_hostAOS_Force;

static float *g_dptrAOS_PosMass;
static float *g_dptrAOS_Force;


// Buffer to hold the golden version of the forces, used for comparison
// Along with timing results, we report the maximum relative error with
// respect to this array.
static float *g_hostAOS_Force_Golden;

float *g_hostSOA_Pos[3];
float *g_hostSOA_Force[3];
float *g_hostSOA_Mass;
float *g_hostSOA_InvMass;

static size_t g_N;

static float g_softening = 0.1f;
static float g_damping = 0.995f;
static float g_dt = 0.016f;

template<typename T>
static T
relError( T a, T b )
{
    if ( a == b ) return 0.0f;
    T relErr = (a-b)/b;
    // Manually take absolute value
    return (relErr<0.0f) ? -relErr : relErr;
}

#include "nbody_CPU_AOS.h"
#include "nbody_CPU_AOS_tiled.h"
#include "nbody_CPU_SOA.h"
#include "nbody_CPU_SOA_tiled.h"
#include "nbody_CPU_SIMD.h"

#ifndef NO_CUDA
#include "nbody_GPU_AOS.cuh"
#include "nbody_GPU_AOS_const.cuh"
#include "nbody_GPU_AOS_tiled.cuh"
#include "nbody_GPU_AOS_tiled_const.cuh"
//#include "nbody_GPU_SOA_tiled.cuh"
#include "nbody_GPU_Shuffle.cuh"
#include "nbody_GPU_Atomic.cuh"
#endif

static void
integrateGravitation_AOS( float *ppos, float *pvel, float *pforce, float dt, float damping, size_t N )
{
    for ( size_t i = 0; i < N; i++ ) {
        const int index = 4*i;
        const int indexForce = 3*i;

        float pos[3], vel[3], force[3];
        pos[0] = ppos[index+0];
        pos[1] = ppos[index+1];
        pos[2] = ppos[index+2];
        float invMass = pvel[index+3];

        vel[0] = pvel[index+0];
        vel[1] = pvel[index+1];
        vel[2] = pvel[index+2];

        force[0] = pforce[indexForce+0];
        force[1] = pforce[indexForce+1];
        force[2] = pforce[indexForce+2];

        // acceleration = force / mass;
        // new velocity = old velocity + acceleration * deltaTime
        vel[0] += (force[0] * invMass) * dt;
        vel[1] += (force[1] * invMass) * dt;
        vel[2] += (force[2] * invMass) * dt;

        vel[0] *= damping;
        vel[1] *= damping;
        vel[2] *= damping;

        // new position = old position + velocity * deltaTime
        pos[0] += vel[0] * dt;
        pos[1] += vel[1] * dt;
        pos[2] += vel[2] * dt;

        ppos[index+0] = pos[0];
        ppos[index+1] = pos[1];
        ppos[index+2] = pos[2];

        pvel[index+0] = vel[0];
        pvel[index+1] = vel[1];
        pvel[index+2] = vel[2];
    }
}

static enum nbodyAlgorithm_enum g_Algorithm;

//
// g_maxAlgorithm is used to determine when to rotate g_Algorithm back to CPU_AOS
// If CUDA is present, it depends on SM version
//
// The shuffle and tiled implementations are SM 3.0 only.
//
// The CPU and GPU algorithms must be contiguous, and the logic in main() to
// initialize this value must be modified if any new algorithms are added.
//
static enum nbodyAlgorithm_enum g_maxAlgorithm;
static int g_bCrossCheck = 1;
static bool g_bUseSIMDForCrossCheck = false;
static int g_bNoCPU = 0;

static bool
ComputeGravitation(
    float *ms,
    float *maxRelError,
    nbodyAlgorithm_enum algorithm,
    bool bCrossCheck )
{
    hipError_t status;
    bool bSOA = false;

    // AOS -> SOA data structures in case we are measuring SOA performance
    for ( size_t i = 0; i < g_N; i++ ) {
        g_hostSOA_Pos[0][i]  = g_hostAOS_PosMass[4*i+0];
        g_hostSOA_Pos[1][i]  = g_hostAOS_PosMass[4*i+1];
        g_hostSOA_Pos[2][i]  = g_hostAOS_PosMass[4*i+2];
        g_hostSOA_Mass[i]    = g_hostAOS_PosMass[4*i+3];
        g_hostSOA_InvMass[i] = 1.0f / g_hostSOA_Mass[i];
    }

    if ( bCrossCheck ) {
#ifdef HAVE_SIMD
        if ( g_bUseSIMDForCrossCheck ) {
            ComputeGravitation_SIMD(
                            g_hostSOA_Force,
                            g_hostSOA_Pos,
                            g_hostSOA_Mass,
                            g_softening*g_softening,
                            g_N );
        } else
#endif
        {
            ComputeGravitation_SOA(
                            g_hostSOA_Force,
                            g_hostSOA_Pos,
                            g_hostSOA_Mass,
                            g_softening*g_softening,
                            g_N );
        }
        for ( size_t i = 0; i < g_N; i++ ) {
            g_hostAOS_Force_Golden[3*i+0] = g_hostSOA_Force[0][i];
            g_hostAOS_Force_Golden[3*i+1] = g_hostSOA_Force[1][i];
            g_hostAOS_Force_Golden[3*i+2] = g_hostSOA_Force[2][i];
        }
    }

    /* Reset the force values so we know the function tested did work. */
    memset(g_hostAOS_Force,    0, g_N * sizeof(float) * 3);
    memset(g_hostSOA_Force[0], 0, g_N * sizeof(float));
    memset(g_hostSOA_Force[1], 0, g_N * sizeof(float));
    memset(g_hostSOA_Force[2], 0, g_N * sizeof(float));

    // CPU->GPU copies in case we are measuring GPU performance
    if ( g_bCUDAPresent ) {
        CUDART_CHECK( hipMemcpyAsync(
            g_dptrAOS_PosMass,
            g_hostAOS_PosMass,
            4*g_N*sizeof(float),
            hipMemcpyHostToDevice ) );
    }

    switch ( algorithm ) {
        case CPU_AOS:
            *ms = ComputeGravitation_AOS(
                g_hostAOS_Force,
                g_hostAOS_PosMass,
                g_softening*g_softening,
                g_N );
            break;
        case CPU_AOS_tiled:
            *ms = ComputeGravitation_AOS_tiled(
                g_hostAOS_Force,
                g_hostAOS_PosMass,
                g_softening*g_softening,
                g_N );
            break;
        case CPU_SOA:
            *ms = ComputeGravitation_SOA(
                g_hostSOA_Force,
                g_hostSOA_Pos,
                g_hostSOA_Mass,
                g_softening*g_softening,
                g_N );
            bSOA = true;
            break;
        case CPU_SOA_tiled:
            *ms = ComputeGravitation_SOA_tiled(
                g_hostSOA_Force,
                g_hostSOA_Pos,
                g_hostSOA_Mass,
                g_softening*g_softening,
                g_N );
            bSOA = true;
            break;
#ifdef HAVE_SIMD
        case CPU_SIMD:
            *ms = ComputeGravitation_SIMD(
                g_hostSOA_Force,
                g_hostSOA_Pos,
                g_hostSOA_Mass,
                g_softening*g_softening,
                g_N );
            bSOA = true;
            break;
#endif
#ifndef NO_CUDA
        case GPU_AOS:
            *ms = ComputeGravitation_GPU_AOS(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_AOS_tiled:
            *ms = ComputeGravitation_GPU_AOS_tiled(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_AOS_tiled_const:
            *ms = ComputeGravitation_GPU_AOS_tiled_const(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
#if 0
// commented out - too slow even on SM 3.0
        case GPU_Atomic:
            CUDART_CHECK( hipMemset( g_dptrAOS_Force, 0, 3*sizeof(float) ) );
            *ms = ComputeGravitation_GPU_Atomic(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
#endif
        case GPU_Shared:
            CUDART_CHECK( hipMemset( g_dptrAOS_Force, 0, 3*g_N*sizeof(float) ) );
            *ms = ComputeGravitation_GPU_Shared(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_Const:
            CUDART_CHECK( hipMemset( g_dptrAOS_Force, 0, 3*g_N*sizeof(float) ) );
            *ms = ComputeNBodyGravitation_GPU_AOS_const(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_Shuffle:
            CUDART_CHECK( hipMemset( g_dptrAOS_Force, 0, 3*g_N*sizeof(float) ) );
            *ms = ComputeGravitation_GPU_Shuffle(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case multiGPU:
            memset( g_hostAOS_Force, 0, 3*g_N*sizeof(float) );
            *ms = ComputeGravitation_multiGPU(
                g_hostAOS_Force,
                g_hostAOS_PosMass,
                g_softening*g_softening,
                g_N );
            break;
#endif
        default:
            fprintf(stderr, "Unrecognized algorithm index: %d\n", algorithm);
            abort();
    }

    // SOA -> AOS
    if ( bSOA ) {
        for ( size_t i = 0; i < g_N; i++ ) {
            g_hostAOS_Force[3*i+0] = g_hostSOA_Force[0][i];
            g_hostAOS_Force[3*i+1] = g_hostSOA_Force[1][i];
            g_hostAOS_Force[3*i+2] = g_hostSOA_Force[2][i];
        }
    }

    *maxRelError = 0.0f;
    if ( bCrossCheck ) {
        float max = 0.0f;
        for ( size_t i = 0; i < 3*g_N; i++ ) {
            float err = relError( g_hostAOS_Force[i], g_hostAOS_Force_Golden[i] );
            if ( err > max ) {
                max = err;
            }
        }
        *maxRelError = max;
    }

    integrateGravitation_AOS(
        g_hostAOS_PosMass,
        g_hostAOS_VelInvMass,
        g_hostAOS_Force,
        g_dt,
        g_damping,
        g_N );
    return true;
Error:
    return false;
}

static workerThread *g_GPUThreadPool;
int g_numGPUs;

struct gpuInit_struct
{
    int iGPU;

    hipError_t status;
};

static void
initializeGPU( void *_p )
{
    hipError_t status;

    gpuInit_struct *p = (gpuInit_struct *) _p;
    CUDART_CHECK( hipSetDevice( p->iGPU ) );
    CUDART_CHECK( hipSetDeviceFlags( hipDeviceMapHost ) );
    CUDART_CHECK( hipFree(0) );
Error:
    p->status = status;
}

static void usage(const char *argv0)
{
    printf( "Usage: nbody --bodies=N [--gpus=N] [--no-cpu] [--no-crosscheck] [--cycle-after=N] [--iterations=N]\n" );
    printf( "    --bodies is multiplied by 1024 (default is 16)\n" );
    printf( "    By default, the app checks results against a CPU implementation; \n" );
    printf( "    disable this behavior with --no-crosscheck.\n" );
    printf( "    The CPU implementation may be disabled with --no-cpu.\n" );
    printf( "    --no-cpu implies --no-crosscheck.\n\n" );
    printf( "    --iterations specifies a fixed number of iterations to execute\n" );
    printf( "    --cycle-after specifies the number of iterations before rotating\n" );
    printf( "                  to the next available algorithm\n" );
}

int
main( int argc, char *argv[] )
{
    hipError_t status;
    // kiloparticles
    int kParticles = 16, kMaxIterations = 0, kCycleAfter = 0;

    static const struct option cli_options[] = {
        { "bodies", required_argument, NULL, 'b' },
        { "gpus", required_argument, NULL, 'g' },
        { "no-cpu", no_argument, &g_bNoCPU, 1 },
        { "no-crosscheck", no_argument, &g_bCrossCheck, 0 },
        { "iterations", required_argument, NULL, 'i' },
        { "cycle-after", required_argument, NULL, 'c' },
        { "help", no_argument, NULL, 'h' },
        { NULL, 0, NULL, 0 }
    };

    status = hipGetDeviceCount( &g_numGPUs );
    if (status != hipSuccess)
        g_numGPUs = 0;

    while (1) {
        int option = getopt_long(argc, argv, "n:i:c:", cli_options, NULL);

        if (option == -1)
            break;

        switch (option) {
        case 'c':
            {
                int v;
                if (sscanf(optarg, "%d", &v) != 1) {
                    fprintf(stderr, "ERROR: Couldn't parse integer argument for '--cycle-after'\n");
                    return 1;
                }
                if (v < 1) {
                    fprintf(stderr, "ERROR: Requested cycle size less than 1\n");
                    return 1;
                }
                kCycleAfter = v;
            }
            break;
        case 'i':
            {
                int v;
                if (sscanf(optarg, "%d", &v) != 1) {
                    fprintf(stderr, "ERROR: Couldn't parse integer argument for '--iterations'\n");
                    return 1;
                }
                if (v < 1) {
                    fprintf(stderr, "ERROR: Requested number of iterations less than 1\n");
                    return 1;
                }
                kMaxIterations = v;
            }
            break;
        case 'b':
            {
                int v;
                if (sscanf(optarg, "%d", &v) != 1) {
                    fprintf(stderr, "ERROR: Couldn't parse integer argument for '--bodies'\n");
                    return 1;
                }
                if (v < 1) {
                    printf("ERROR: Requested number of bodies less than 1");
                    return 1;
                }
                kParticles = v;
            }
            break;
        case 'g':
            {
                int v;
                if (sscanf(optarg, "%d", &v) != 1) {
                    fprintf(stderr, "ERROR: Couldn't parse integer argument for '--gpus'\n");
                    return 1;
                }
                if (v < 1) {
                    if (g_numGPUs > 0)
                        fprintf(stderr, "Requested number of GPUs less than 1, disabling GPU algorithms.\n");
                    g_numGPUs = 0;
                    break;
                }
                if (v > g_numGPUs) {
                    fprintf(stderr, "Requested %d GPUs, but only have %d, using all available GPUs.\n",
                            v, g_numGPUs);
                    break;
                }
                g_numGPUs = v;
            }
            break;
        case 'h':
        case '?':
            usage(argv[0]);
            return 1;
        }
    }

    // for reproducible results for a given N
    srand(7);

    g_bCUDAPresent = g_numGPUs > 0;
    if ( g_bCUDAPresent ) {
        hipDeviceProp_t prop;
        CUDART_CHECK( hipGetDeviceProperties( &prop, 0 ) );
        g_bSM30Present = prop.major >= 3;
    }

    if ( g_bNoCPU && ! g_bCUDAPresent ) {
        fprintf(stderr, "ERROR: --no-cpu specified, but no CUDA present\n" );
        exit(1);
    }

    if ( g_numGPUs ) {
        g_GPUThreadPool = new workerThread[g_numGPUs];
        for ( int i = 0; i < g_numGPUs; i++ ) {
            if ( ! g_GPUThreadPool[i].initialize( ) ) {
                fprintf( stderr, "Error initializing thread pool\n" );
                return 1;
            }
        }
        for ( int i = 0; i < g_numGPUs; i++ ) {
            gpuInit_struct initGPU = {i};
            g_GPUThreadPool[i].delegateSynchronous(
                initializeGPU,
                &initGPU );
            if ( hipSuccess != initGPU.status ) {
                fprintf( stderr, "Initializing GPU %d failed "
                    " with %d (%s)\n",
                    i,
                    initGPU.status,
                    hipGetErrorString( initGPU.status ) );
                return 1;
            }
        }
    }

    if ( g_bNoCPU ) {
        g_bCrossCheck = false;
    }

    g_N = kParticles * 1024;

    printf( "Running simulation with %d particles, crosscheck %s, CPU %s\n", (int) g_N,
        g_bCrossCheck ? "enabled" : "disabled",
        g_bNoCPU ? "disabled" : "enabled" );

#if defined(HAVE_SIMD)
    g_maxAlgorithm = CPU_SIMD;
#else
    g_maxAlgorithm = CPU_SOA_tiled;
#endif
    g_Algorithm = g_bCUDAPresent ? GPU_AOS : CPU_SOA;
    if ( g_bCUDAPresent || g_bNoCPU ) {
        // max algorithm is different depending on whether SM 3.0 is present
        g_maxAlgorithm = g_bSM30Present ? GPU_AOS_tiled_const : multiGPU;
    }

    if ( g_bCUDAPresent ) {
        hipDeviceProp_t propForVersion;

        CUDART_CHECK( hipSetDeviceFlags( hipDeviceMapHost ) );
        CUDART_CHECK( hipGetDeviceProperties( &propForVersion, 0 ) );
        if ( propForVersion.major < 3 ) {
            // Only SM 3.x supports shuffle and fast atomics, so we cannot run
            // some algorithms on this board.
            g_maxAlgorithm = multiGPU;
        }

        CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_PosMass, 4*g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        for ( size_t i = 0; i < 3; i++ ) {
            CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_Pos[i], g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
            CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_Force[i], g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        }
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_Force, 3*g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_Force_Golden, 3*g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_VelInvMass, 4*g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_Mass, g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_InvMass, g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );

        CUDART_CHECK( hipMalloc( &g_dptrAOS_PosMass, 4*g_N*sizeof(float) ) );
        CUDART_CHECK( hipMalloc( (void **) &g_dptrAOS_Force, 3*g_N*sizeof(float) ) );
    }
    else {
        g_hostAOS_PosMass = new float[4*g_N];
        for ( size_t i = 0; i < 3; i++ ) {
            g_hostSOA_Pos[i] = new float[g_N];
            g_hostSOA_Force[i] = new float[g_N];
        }
        g_hostSOA_Mass = new float[g_N];
        g_hostAOS_Force = new float[3*g_N];
        g_hostAOS_Force_Golden = new float[3*g_N];
        g_hostAOS_VelInvMass = new float[4*g_N];
        g_hostSOA_Mass = new float[g_N];
        g_hostSOA_InvMass = new float[g_N];
    }

    randomUnitBodies( g_hostAOS_PosMass, g_hostAOS_VelInvMass, g_N );
    for ( size_t i = 0; i < g_N; i++ ) {
        g_hostSOA_Mass[i] = g_hostAOS_PosMass[4*i+3];
        g_hostSOA_InvMass[i] = 1.0f / g_hostSOA_Mass[i];
    }

#if 0
    // gather performance data over GPU implementations
    // for different problem sizes.

    printf( "kBodies\t" );
    for ( int algorithm = GPU_AOS;
              algorithm < sizeof(rgszAlgorithmNames)/sizeof(rgszAlgorithmNames[0]);
              algorithm++ ) {
        printf( "%s\t", rgszAlgorithmNames[algorithm] );
    }
    printf( "\n" );

    for ( int kBodies = 3; kBodies <= 96; kBodies += 3 ) {

        g_N = 1024*kBodies;

        printf( "%d\t", kBodies );

        for ( int algorithm = GPU_AOS;
                  algorithm < sizeof(rgszAlgorithmNames)/sizeof(rgszAlgorithmNames[0]);
                  algorithm++ ) {
            float sum = 0.0f;
            const int numIterations = 10;
            for ( int i = 0; i < numIterations; i++ ) {
                float ms, err;
                if ( ! ComputeGravitation( &ms, &err, (nbodyAlgorithm_enum) algorithm, g_bCrossCheck ) ) {
                        fprintf( stderr, "Error computing timestep\n" );
                        exit(1);
                }
                sum += ms;
            }
            sum /= (float) numIterations;

            double interactionsPerSecond = (double) g_N*g_N*1000.0f / sum;
            if ( interactionsPerSecond > 1e9 ) {
                printf ( "%.2f\t", interactionsPerSecond/1e9 );
            }
            else {
                printf ( "%.3f\t", interactionsPerSecond/1e9 );
            }
        }
        printf( "\n" );
    }
    return 0;
#endif
    {
        int kIterations = 0;
        bool bStop = false;
        while ( ! bStop ) {
            float ms, err;

            if ( ! ComputeGravitation( &ms, &err, g_Algorithm, g_bCrossCheck ) ) {
                fprintf( stderr, "Error computing timestep\n" );
                exit(1);
            }
            double interactionsPerSecond = (double) g_N*g_N*1000.0f / ms,
                   flops = (g_N * g_N * (3 + 6 + 4 + 1 + 6)) * 1000.0f / ms;
            if ( interactionsPerSecond > 1e9 ) {
                printf ( "\r%13s: %8.2f ms = %8.3fx10^9 interactions/s (%9.2lf GFLOPS)",
                    rgszAlgorithmNames[g_Algorithm],
                    ms,
                    interactionsPerSecond/1e9,
                    flops * 1e-9 );
            }
            else {
                printf ( "\r%13s: %8.2f ms = %8.3fx10^6 interactions/s (%9.2lf GFLOPS)",
                    rgszAlgorithmNames[g_Algorithm],
                    ms,
                    interactionsPerSecond/1e6,
                    flops * 1e-9 );
            }
            if (g_bCrossCheck)
                printf( " (Rel. error: %E)\n", err );
            else
                printf( "\n" );

            kIterations++;
            if (kMaxIterations) {
                int kIterationRatio = kCycleAfter * (g_maxAlgorithm + 1);
                if (!kIterationRatio)
                    kIterationRatio = 1;
                if (kIterations / kIterationRatio >= kMaxIterations) {
                    bStop = true;
                }
            }
            if (kCycleAfter && kIterations % kCycleAfter == 0) {
                g_Algorithm = (enum nbodyAlgorithm_enum) (g_Algorithm+1);
                if ( g_Algorithm > g_maxAlgorithm ) {
                    g_Algorithm = g_bNoCPU ? GPU_AOS : CPU_AOS;
                }
            }
            if ( kbhit() ) {
                char c = getch();
                switch ( c ) {
                    case ' ':
                        g_Algorithm = (enum nbodyAlgorithm_enum) (g_Algorithm+1);
                        if ( g_Algorithm > g_maxAlgorithm ) {
                            g_Algorithm = g_bNoCPU ? GPU_AOS : CPU_AOS;
                        }
                        break;
                    case 'q':
                    case 'Q':
                        bStop = true;
                        break;
                }

            }
        }
    }

    return 0;
Error:
    if ( hipSuccess != status ) {
        printf( "CUDA Error: %s\n", hipGetErrorString( status ) );
    }
    return 1;
}

/* vim: set ts=4 sts=4 sw=4 et: */
